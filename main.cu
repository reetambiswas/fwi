#include "hip/hip_runtime.h"
#include"global.h"
#include"header.h"


using namespace std;



int main()
{
	pad=60;
	O=10;

	//Define Dimensions of model
	dimMod[0]=122;dimMod[1]=384;

	NX=dimMod[1]+2*pad;
	NZ=dimMod[0]+2*pad;
	NZ1=dimMod[0]+2*pad;


	

	//Parameters for the modelling
	float t_n=3;
	float f=15;
	dt=0.001;
	dx=10;
	dz=dx;
	

	int nt=(int)t_n/dt;

	//Define File Names
	// std::string veloFn="marmousi.dat";
	// std::string veloFn="marmsmooth.dat";
	// std::string veloFn="marmhard.dat";
	std::string veloFn="homogenous.velo";
	// std::string sourceFn="wavelet2.dat";
	
	

	float velo1[dimMod[0]*dimMod[1]],
	field1[NX*NZ],//[dimMod[0]*dimMod[1]],
	field2[NX*NZ],//[dimMod[0]*dimMod[1]],
	velocity[NZ*NX];
	
	consVec(field1, NX*NZ,/*dimMod[0]*dimMod[1]*/ 0);
	consVec(field2, NX*NZ,/*dimMod[0]*dimMod[1]*/ 0);

	
	ofstream myFile ("wavefield1.bin", ios::out | ios::binary);
	
	
	dimW[0]=nt;dimW[1]=1;
	float wavelet[dimW[0]*dimW[1]];
	
	
	
	//Read files for velocity model, wavelet etc.
	readASCIIFile((char*)veloFn.c_str(),dimMod, velo1);
	matrixPadding(velocity,velo1, dimMod[0], dimMod[1],pad);

	// VectoFileWrite(velocity, NX*NZ, "outVelocity.dat");
	// exit(1);
	dx=max(velo1,dimMod)/f/6;
	//readASCIIFile((char*)sourceFn.c_str(),dimW, wavelet);
	
	// source_x=(dimMod[1]/2)*dx;//dimMod[1]/2*dx;
	// source_z=(dimMod[0]/2)*dx;//(30)*dx,
	source_x=(dimMod[1]/2+pad)*dx;
	source_z=(10+pad)*dx,
	bc=1;
	

	// float C[11]={0.565794,-6.261905,31.544643,-95.523810,193.361111,-275.080000,281.291667,-207.650794,109.303571,-38.579365,7.029087};
	//float C[11]={7.029087,-38.579365,109.303571,-207.650794,281.291667,-275.080000,193.361111,-95.523810,31.544643,-6.261905,0.565794};
	float C[11]={0.000317,-0.004960,0.039683,-0.238095,1.666667,-2.927222,1.666667,-0.238095,0.039683,-0.004960,0.000317};
	// float CC[11];
	
	clock_t t1,t2,t3,t4;
	
	

	printf("Position of Source:\nX:%f\nZ:%f\n",source_x,source_z);
	

	double stability=(double)(dt/dx)*max(velo1,dimMod);
	printf("Stability=%f\n",(float)stability);
	
	if(stability>1/sqrt(2))
	{
		printf("Grid not stable! Check the parameters and run again!\n");
		exit(1);		
	}	
	
	
	
	
	//VectoFileWrite(field2, dimMod[0]*dimMod[1], "field2.dat");
	t1=clock();
	//Initializing Cuda Device
	int numDevs= 0;
	hipGetDeviceCount(&numDevs);
	//For now using only one device
	printf("Number of Device:%d\n",numDevs);
	int deviceID=0;
	hipSetDevice(deviceID );
	int cDeviceID=0;
	hipGetDevice(&cDeviceID );
	printf("Current Active Device ID:%d\n",cDeviceID);
	
	dim3 blockDim(32,32);
	//dim3 gridDim((int)ceil(dimMod[1]/32)+1,(int)ceil(dimMod[0]/32)+1);
	dim3 gridDim(20,20);


	printf("GridDim:(%d,%d)\nBlockDim(%d,%d)\n",gridDim.x,gridDim.y,blockDim.x,blockDim.y);
	
	t2=clock();
	//Creating Device variables in cuda devices
	float *d_field1, *d_field2, *d_wavelet, *source_grid, *d_velocity, *d_wave_propagate_t, 
	*d_laplace_temp, *d_laplace, *d_C, *d_correctField, *d_temp1, *d_G, *d_cerjanMatrix;
	hipError_t cudaStatus;
	
	cudaStatus=hipMalloc((void**)&d_field1,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);
	
	
	cudaStatus=hipMalloc((void**)&d_field2,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_temp1,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);
	
	cudaStatus=hipMalloc((void**)&d_wavelet,dimW[0]*dimW[1]*sizeof(float));
	cudaCheck(cudaStatus);
	
	cudaStatus=hipMalloc((void**)&source_grid,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_correctField,dimMod[0]*dimMod[1]*sizeof(float));
	cudaCheck(cudaStatus);
	
	// cudaStatus=hipMalloc((void**)&d_velocity,dimMod[0]*dimMod[1]*sizeof(float));
	// cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_velocity,NX*NZ*sizeof(float));
	cudaCheck(cudaStatus);
	
	cudaStatus=hipMalloc((void**)&d_wave_propagate_t,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);
	
	cudaStatus=hipMalloc((void**)&d_laplace_temp,(NZ/*dimMod[0]*/+2*O)*(NX/*dimMod[1]*/+2*O)*sizeof(float));
	cudaCheck(cudaStatus);
	
	cudaStatus=hipMalloc((void**)&d_laplace,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_cerjanMatrix,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float));
	cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_C,(O+1)*sizeof(float));
	cudaCheck(cudaStatus);

	cudaStatus=hipMalloc((void**)&d_G,(O+pad)*sizeof(float));
	cudaCheck(cudaStatus);
	
	//Generating Wavelet

	rickerWavelet<<<gridDim,blockDim>>>( d_wavelet, f, dimW[0]*dimW[1], dt);
	cudaStatus = hipMemcpy(wavelet, d_wavelet , dimW[0]*dimW[1]*sizeof(float), hipMemcpyDeviceToHost);
	cudaCheck(cudaStatus);
	VectoFileWrite(wavelet, dimW[0]*dimW[1], "outWavelet.dat");
	

	//Constructing the initial fields by placing the source in correct position		
	construct_source(field1,field2,wavelet[0]);


	//Copying Data to Device
	cudaStatus = hipMemcpy(d_field1, field1 , NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);
	
	cudaStatus = hipMemcpy(d_field2, field2 , NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);

	cudaStatus = hipMemcpy(d_wavelet, wavelet , dimW[0]*dimW[1]*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);
	
	cudaStatus = hipMemcpy(source_grid, field2 , NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);
	
	cudaStatus = hipMemcpy(d_velocity, velocity , NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);

	cudaStatus = hipMemcpy(d_C, C , (O+1)*sizeof(float), hipMemcpyHostToDevice);
	cudaCheck(cudaStatus);

	/*cudaStatus = hipMemcpy(CC, d_C , O*sizeof(float), hipMemcpyDeviceToHost);
	cudaCheck(cudaStatus);
	VectoFileWrite(CC, O, "CC.dat");
	*/

	// Calculating Cerjan Boundary Condition Coefficients
	calculateCerjanCoeff<<<gridDim,blockDim>>>(pad, d_G, 60);

	cerjanMatrix<<<gridDim,blockDim>>>(d_cerjanMatrix, d_G, NZ,NX, pad);

	float CM[NX*NZ];
	cudaStatus = hipMemcpy(CM, d_cerjanMatrix , (NZ*NX)*sizeof(float), hipMemcpyDeviceToHost);
	cudaCheck(cudaStatus);
	VectoFileWrite(CM, NX*NZ, "outCerjanMatrix.dat");

	float G[pad+O];
	cudaStatus = hipMemcpy(G, d_G , (pad)*sizeof(float), hipMemcpyDeviceToHost);
	cudaCheck(cudaStatus);

	VectoFileWrite(G, pad, "outCerjan.dat");
	// exit(1);

	int size=dimMod[0]*dimMod[1]; //(dimMod[0])*(dimMod[1]);
	float check[size];
	// int size1=(dimMod[0]+2*O)*(dimMod[1]+2*O);
	// float check1[size1];
	// cudaStatus = hipMemcpy(check, d_velocity , size*sizeof(float), hipMemcpyDeviceToHost);
	// if(cudaStatus!=hipSuccess)
	// {
		// printf("Cuda couldn't allocated! Error no.:%d\n",(int)cudaStatus);
	// }
	// VectoFileWrite(check, size, "check.dat");
	t3=clock();
	printf("Running Loop!\n");
	//char buffer[32];
	for(int step=1; step<=nt; step++)
	{
		//Wave Propagation
		// fdm_acoustic(d_velocity,d_field1,d_field2,bc,gridDim,blockDim);
		
		//Calculate Laplacian
		// myCudaMemset  (  d_laplace_temp,0.0, (dimMod[0]+2*O),(dimMod[1]+2*O)  );      
		myCudaMemset<<<gridDim,blockDim>>>(d_laplace_temp, 0.0, (NZ/*dimMod[0]*/+2*O), (NX/*dimMod[1]*/+2*O));

		//myCudaMemset<<<gridDim,blockDim>>>(  d_field2,1.0, (dimMod[0]),(dimMod[1]));      
		

		// cudaStatus = hipMemcpy(check, d_field2 , size*sizeof(float), hipMemcpyDeviceToHost);
		// cudaCheck(cudaStatus);
		// VectoFileWrite(check, size, "field2.dat");
		
		
		//wavefield Transfer
		wavefieldTransfer<<<gridDim,blockDim>>>(d_laplace_temp,d_field2,NZ/*dimMod[0]*/,NX/*dimMod[1]*/,O);
		// cudaStatus = hipMemcpy(check1, d_laplace_temp , size1*sizeof(float), hipMemcpyDeviceToHost);
		// cudaCheck(cudaStatus);
		// VectoFileWrite(check1, size1, "check1.dat");
		// exit(1);
		
		//Propagating waves
		calculateLaplace<<<gridDim,blockDim>>>(d_laplace, d_wave_propagate_t, d_laplace_temp, d_velocity,
												d_field1,d_field2,dt,dx, NZ/*dimMod[0]*/,NX/*dimMod[1]*/, bc, O, d_C);
		// cudaStatus = hipMemcpy(check, d_laplace , size*sizeof(float), hipMemcpyDeviceToHost);
		// cudaCheck(cudaStatus);
		// VectoFileWrite(check, size, "check.dat");
		//exit(1);
			
		//ABC
		 ABC_inner<<<gridDim,blockDim>>>(d_velocity,d_field1,d_field2,d_wave_propagate_t,dx,dt,bc,NZ/*dimMod[0]*/,NX/*dimMod[1]*/);
		//ABC_outer<<<gridDim,blockDim>>>(d_velocity,d_field1,d_field2,d_wave_propagate_t,dx,dt,bc,dimMod[0],dimMod[1]);
		// BadBoundaryCondition makes everything in the extra region zero
		// myCudaMemset<<<gridDim,blockDim>>>(d_temp1, 0.0, NZ/*dimMod[0]*/, NX);
		// badBoundaryCondition<<<gridDim,blockDim>>>(d_wave_propagate_t,d_temp1,dimMod[0],dimMod[1],pad);
		// cudaStatus=hipMemcpy(d_wave_propagate_t,d_temp1,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float),hipMemcpyDeviceToDevice);
		// cudaCheck(cudaStatus);

		// myCudaMemset<<<gridDim,blockDim>>>(d_temp1, 0.0, NZ/*dimMod[0]*/, NX);
		// badBoundaryCondition<<<gridDim,blockDim>>>(d_field1,d_temp1,dimMod[0],dimMod[1],pad);
		// cudaStatus=hipMemcpy(d_field1,d_temp1,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float),hipMemcpyDeviceToDevice);
		// cudaCheck(cudaStatus);

		// myCudaMemset<<<gridDim,blockDim>>>(d_temp1, 0.0, NZ/*dimMod[0]*/, NX);
		// badBoundaryCondition<<<gridDim,blockDim>>>(d_field2,d_temp1,dimMod[0],dimMod[1],pad);
		// cudaStatus=hipMemcpy(d_field2,d_temp1,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float),hipMemcpyDeviceToDevice);
		// cudaCheck(cudaStatus);

		// cerjanBoundaryCondition<<<gridDim,blockDim>>>( d_wave_propagate_t,  d_field1,  d_field2, d_cerjanMatrix, NZ, NX);

			
			
			
		//field1=field2;
		cudaStatus=hipMemcpy(d_field1,d_field2,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float),hipMemcpyDeviceToDevice);
			cudaCheck(cudaStatus);
			
			
			
			if(step+1<dimW[0])
			{
			add_source<<<gridDim,blockDim>>>(d_wave_propagate_t,d_field2,source_grid,wavelet[step+1],NZ/*dimMod[0]*/,NX/*dimMod[1]*/);
				
			}
			else
			{
			cudaStatus=hipMemcpy(d_field2,d_wave_propagate_t,NX*NZ/*dimMod[0]*dimMod[1]*/*sizeof(float),hipMemcpyDeviceToDevice);
				cudaCheck(cudaStatus);			
			}
			
			if(remainderf((float)step,1)==0)
			{
				extractCorrectRegion<<<gridDim,blockDim>>>(d_correctField,d_field2,dimMod[0],dimMod[1],pad);
				cudaStatus=hipMemcpy(check,d_correctField,dimMod[0]*dimMod[1]*sizeof(float),hipMemcpyDeviceToHost);
				cudaCheck(cudaStatus);
			// snprintf(buffer, sizeof(char) * 32, "file%d.txt", step);
			// VectoFileWrite(check, size, buffer);
				myFile.write ((char*)check, size*sizeof(float));
			}
			
		}

		
		myFile.close();
		t4=clock();

		printf("Total Execution Time:%f\n",((float)t4-(float)t1)/CLOCKS_PER_SEC);
		printf("Cuda Device Query Execution Time:%f\n",((float)t2-(float)t1)/CLOCKS_PER_SEC);
		printf("Cuda Malloc Execution Time:%f\n",((float)t3-(float)t2)/CLOCKS_PER_SEC);
		printf("Cuda Loop Execution Time:%f\n",((float)t4-(float)t3)/CLOCKS_PER_SEC);

		hipFree(d_field1);
		hipFree(d_field2);
		hipFree(d_wavelet);
		hipFree(source_grid);
		hipFree(d_velocity);
		hipFree(d_wave_propagate_t);
		hipFree(d_laplace_temp);
		hipFree(d_laplace);
		
		
		
		


		return 0;
		
		
	}
