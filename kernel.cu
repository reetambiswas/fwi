#include "hip/hip_runtime.h"
#include"global.h"
#include"header.h"

__shared__  int d_dimW[2];


/*__global__ void fdm_acoustic(dx,dt,d_velocity,d_field1,d_field2,bc);
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	
}*/

__device__ int ind(int i, int j, int m, int n)
{
	return i+j*m;
}


__global__ void wavefieldTransfer(float* d_laplace_temp, float* d_field2, int m, int n, int O)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	
	
	if (row >= m || col >= n)return;
	
	
	d_laplace_temp[ind(row+O,col+O,m+2*O,n+2*O)]=d_field2[ind(row,col,m,n)];
	
	
}


__global__ void myCudaMemset(float* A, float val, int m, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	
	
	if (row >= m || col >= n)return;


	A[ind(row,col,m,n)]=val;

	

}




__global__ void calculateLaplace(float* d_laplace,float* d_wave_propagate_t, float* A /*d_laplace_temp*/, float* V,float* d_field1,  float* d_field2, float dt, float dx, int m, int n, float bc, int O, float* C)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row >= m || col >= n)return;

	//O=O+1;

	//if(col>=0 && col <11) printf("C[%d]=%f\n", col, C[col]);
	// printf("dx=%f",dx);
	
	d_laplace[ind(row,col,m,n)]=((C[0] *A[ind(row+O   ,col+O+5 ,m+2*O,n+2*O)])+
								 (C[1] *A[ind(row+O   ,col+O+4 ,m+2*O,n+2*O)])+
								 (C[2] *A[ind(row+O   ,col+O+3 ,m+2*O,n+2*O)])+
								 (C[3] *A[ind(row+O   ,col+O+2 ,m+2*O,n+2*O)])+
								 (C[4] *A[ind(row+O   ,col+O+1 ,m+2*O,n+2*O)])+
								 (C[5] *A[ind(row+O   ,col+O   ,m+2*O,n+2*O)])+
								 (C[6] *A[ind(row+O   ,col+O-1 ,m+2*O,n+2*O)])+
								 (C[7] *A[ind(row+O   ,col+O-2 ,m+2*O,n+2*O)])+
								 (C[8] *A[ind(row+O   ,col+O-3 ,m+2*O,n+2*O)])+
								 (C[9] *A[ind(row+O   ,col+O-4 ,m+2*O,n+2*O)])+
								 (C[10]*A[ind(row+O   ,col+O-5 ,m+2*O,n+2*O)]))/(dx*dx)
								+
								((C[0] *A[ind(row+O+5 ,col+O   ,m+2*O,n+2*O)])+
								 (C[1] *A[ind(row+O+4 ,col+O   ,m+2*O,n+2*O)])+
								 (C[2] *A[ind(row+O+3 ,col+O   ,m+2*O,n+2*O)])+
								 (C[3] *A[ind(row+O+2 ,col+O   ,m+2*O,n+2*O)])+
								 (C[4] *A[ind(row+O+1 ,col+O   ,m+2*O,n+2*O)])+
								 (C[5] *A[ind(row+O   ,col+O   ,m+2*O,n+2*O)])+
								 (C[6] *A[ind(row+O-1 ,col+O   ,m+2*O,n+2*O)])+
								 (C[7] *A[ind(row+O-2 ,col+O   ,m+2*O,n+2*O)])+
								 (C[8] *A[ind(row+O-3 ,col+O   ,m+2*O,n+2*O)])+
								 (C[9] *A[ind(row+O-4 ,col+O   ,m+2*O,n+2*O)])+
								 (C[10]*A[ind(row+O-5 ,col+O   ,m+2*O,n+2*O)]))/(dx*dx);







	/*(A[ind(row+O+1,col+O,m+2*O,n+2*O)]-2*A[ind(row+O,col+O,m+2*O,n+2*O)]+A[ind(row+O-1,col+O,m+2*O,n+2*O)])/(dx*dx)+
								(A[ind(row+O,col+O+1,m+2*O,n+2*O)]-2*A[ind(row+O,col+O,m+2*O,n+2*O)]+A[ind(row+O,col+O-1,m+2*O,n+2*O)])/(dx*dx);




								/*((C[0] *A[ind(row+O   ,col+O   ,m+2*O,n+2*O)])+
								 (C[1] *A[ind(row+O   ,col+O+1 ,m+2*O,n+2*O)])+
								 (C[2] *A[ind(row+O   ,col+O+2 ,m+2*O,n+2*O)])+
								 (C[3] *A[ind(row+O   ,col+O+3 ,m+2*O,n+2*O)])+
								 (C[4] *A[ind(row+O   ,col+O+4 ,m+2*O,n+2*O)])+
								 (C[5] *A[ind(row+O   ,col+O+5 ,m+2*O,n+2*O)])+
								 (C[6] *A[ind(row+O   ,col+O+6 ,m+2*O,n+2*O)])+
								 (C[7] *A[ind(row+O   ,col+O+7 ,m+2*O,n+2*O)])+
								 (C[8] *A[ind(row+O   ,col+O+8 ,m+2*O,n+2*O)])+
								 (C[9] *A[ind(row+O   ,col+O+9 ,m+2*O,n+2*O)])+
								 (C[10]*A[ind(row+O   ,col+O+10,m+2*O,n+2*O)]))/(dx*dx)
								+
								((C[0] *A[ind(row+O   ,col+O   ,m+2*O,n+2*O)])+
								 (C[1] *A[ind(row+O+1 ,col+O   ,m+2*O,n+2*O)])+
								 (C[2] *A[ind(row+O+2 ,col+O   ,m+2*O,n+2*O)])+
								 (C[3] *A[ind(row+O+3 ,col+O   ,m+2*O,n+2*O)])+
								 (C[4] *A[ind(row+O+4 ,col+O   ,m+2*O,n+2*O)])+
								 (C[5] *A[ind(row+O+5 ,col+O   ,m+2*O,n+2*O)])+
								 (C[6] *A[ind(row+O+6 ,col+O   ,m+2*O,n+2*O)])+
								 (C[7] *A[ind(row+O+7 ,col+O   ,m+2*O,n+2*O)])+
								 (C[8] *A[ind(row+O+8 ,col+O   ,m+2*O,n+2*O)])+
								 (C[9] *A[ind(row+O+9 ,col+O   ,m+2*O,n+2*O)])+
								 (C[10]*A[ind(row+O+10,col+O   ,m+2*O,n+2*O)]))/(dx*dx);




	/*(5.859325396829981f*A[ind(row+O-1,col+O-1,m+O,n+O)]-27.485714285746056*A[ind(row+O,col+O-1,m+O,n+O)]+
		62.100000000097751*A[ind(row+O+1,col+O-1,m+O,n+O)]-89.022222222396508*A[ind(row+O+2,col+O-1,m+O,n+O)]+
		86.375000000196621*A[ind(row+O+3,col+O-1,m+O,n+O)]-56.400000000143436*A[ind(row+O+4,col+O-1,m+O,n+O)]+
		23.811111111177066*A[ind(row+O+5,col+O-1,m+O,n+O)]-5.885714285731734*A[ind(row+O+6,col+O-1,m+O,n+O)]+
		0.648214285716317*A[ind(row+O+7,col+O-1,m+O,n+O)])/(dx*dx);/*+
	(5.859325396829981f*A[ind(row+O-1,col+O-1,m+O,n+O)]-27.485714285746056*A[ind(row+O-1,col+O,m+O,n+O)]+
		62.100000000097751*A[ind(row+O-1,col+O+1,m+O,n+O)]-89.022222222396508*A[ind(row+O-1,col+O+2,m+O,n+O)]+
		86.375000000196621*A[ind(row+O-1,col+O+3,m+O,n+O)]-56.400000000143436*A[ind(row+O-1,col+O+4,m+O,n+O)]+
		23.811111111177066*A[ind(row+O-1,col+O+5,m+O,n+O)]-5.885714285731734*A[ind(row+O-1,col+O+6,m+O,n+O)]+
		0.648214285716317*A[ind(row+O-1,col+O+7,m+O,n+O)])/(dx*dx);*/



//	(A[ind(row+2,col+1,m+2,n+2)]-2*A[ind(row+1,col+1,m+2,n+2)]+A[ind(row,col+1,m+2,n+2)])/(dx*dx)+
//								(A[ind(row+1,col+2,m+2,n+2)]-2*A[ind(row+1,col+1,m+2,n+2)]+A[ind(row+1,col,m+2,n+2)])/(dx*dx);
	
	
	d_wave_propagate_t[ind(row,col,m,n)]=(V[ind(row,col,m,n)]*V[ind(row,col,m,n)])*(dt*dt)*d_laplace[ind(row,col,m,n)]+
										2*d_field2[ind(row,col,m,n)]-d_field1[ind(row,col,m,n)];
										
	
	
	if(bc==1)
	{
		d_wave_propagate_t[ind(row,0,m,n)]=0;
		d_wave_propagate_t[ind(row,n-1,m,n)]=0;
		d_wave_propagate_t[ind(1,col,m,n)]=0;
		d_wave_propagate_t[ind(m-1,col,m,n)]=0;		
	}
	else
	{
		d_wave_propagate_t[ind(row,0,m,n)]=0;
		d_wave_propagate_t[ind(row,n-1,m,n)]=0;
		d_wave_propagate_t[ind(m-1,col,m,n)]=0;		
	}
}


__global__ void add_source(float* d_wave_propagate_t, float* d_field2, float* source_grid,float w, int m, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	
	if (row >= m || col >= n)return;
	
	d_field2[ind(row,col,m,n)]=d_wave_propagate_t[ind(row,col,m,n)]+source_grid[ind(row,col,m,n)]*w;
}

__global__ void ABC_inner(float* V,float* f1, float* f2, float* wave, float dx, float dt, float bc,int m, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row >= m || col >= n)return;
	
	if(bc==1)
	{
		//Applying Bounday condition for Top
		if(row==1 && col<n-6)
		{
			 wave[ind(1,col+3,m,n)] = (2*V[ind(1,col+3,m,n)]*dx*dt*dt)/(dx+V[ind(1,col+3,m,n)]*dt)*
         ((wave[ind(2,col+3,m,n)]/(2*dt*dx) - f1[ind(2,col+3,m,n)]/(2*dx*dt)+
         f1[ind(1,col+3,m,n)]/(2*dt*dx))+
    (-1/(2*dt*dt*V[ind(1,col+3,m,n)]))*
         (-2*f2[ind(1,col+3,m,n)] + f1[ind(1,col+3,m,n)] -2*f2[ind(2,col+3,m,n)]+
         f1[ind(2,col+3,m,n)] + wave[ind(2,col+3,m,n)]) +
    V[ind(1,col+3,m,n)]/(4*dx*dx)*
         (wave[ind(2,col+4,m,n)] + f1[ind(1,col+4,m,n)] + wave[ind(2,col+2,m,n)] -
          2* wave[ind(2,col+3,m,n)] - 2*f1[ind(1,col+3,m,n)] + f1[ind(1,col+2,m,n)]));
		}
	}
	
	//Applying Bounday condition for Bottom
	
	if(row==1 && col<n-6)
	{                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                              
		wave[ind(m-2,col+3,m,n)] = -2.*dx*dt*dt*V[ind(m-2,col+3,m,n)]/(dx+V[ind(m-2,col+3,m,n)]*dt)*
       ((-wave[ind(m-3,col+3,m,n)]/(2*dt*dx) - f1[ind(m-2,col+3,m,n)]/(2*dt*dx) +
       f1[ind(m-3,col+3,m,n)]/(2*dt*dx)) +
  1/(2*dt*dt*V[ind(m-2,col+3,m,n)])*
       (-2*f2[ind(m-2,col+3,m,n)] + f1[ind(m-2,col+3,m,n)] + wave[ind(m-3,col+3,m,n)] - 
       2*f2[ind(m-3,col+3,m,n)] + f1[ind(m-3,col+3,m,n)]) + 
  (-V[ind(m-2,col+3,m,n)]/(4*dx*dx))* 
       (wave[ind(m-3,col+4,m,n)] - 2*wave[ind(m-3,col+3,m,n)] + wave[ind(m-3,col+2,m,n)] + 
       f1[ind(m-2,col+4,m,n)] - 2*f1[ind(m-2,col+3,m,n)] + f1[ind(m-2,col+2,m,n)]));
	}

	//Applying Boundary condition for right hand side
	if(row<m-6 && col==n-1)
	{
		wave[ind(row+3,n-2,m,n)] =  -2*dx*dt*dt*V[ind(row+3,n-2,m,n)]/(dx+V[ind(row+3,n-2,m,n)]*dt)*
       ((-wave[ind(row+3,n-3,m,n)]/(2*dt*dx) - f1[ind(row+3,n-2,m,n)]/(2*dt*dx) + 
       f1[ind(row+3,n-3,m,n)]/(2*dt*dx)) + 
  1/(2*dt*dt*V[ind(row+3,n-2,m,n)])*
       (-2*f2[ind(row+3,n-2,m,n)] + f1[ind(row+3,n-2,m,n)] + wave[ind(row+3,n-3,m,n)] - 
       2*f2[ind(row+3,n-3,m,n)] + f1[ind(row+3,n-3,m,n)]) + 
  (-V[ind(row+3,n-2,m,n)]/(4*dx*dx))* 
       (wave[ind(row+4,n-3,m,n)] - 2*wave[ind(row+3,n-3,m,n)] + wave[ind(row+2,n-3,m,n)] + 
       f1[ind(row+4,n-2,m,n)] - 2*f1[ind(row+3,n-2,m,n)] + f1[ind(row+2,n-2,m,n)]));
	}

	//Applying Boundary condition for left hand side
	if(row<m-6 && col==1)
	{
		wave[ind(row+3,1,m,n)] =(2*V[ind(row+3,1,m,n)]*dx*dt*dt)/(dx+V[ind(row+3,1,m,n)]*dt)*
       ((wave[ind(row+3,2,m,n)]/(2*dt*dx) - f1[ind(row+3,2,m,n)]/(2*dt*dx) +
       f1[ind(row+3,1,m,n)]/(2*dt*dx)) + 
  (-1/(2*dt*dt*V[ind(row+3,1,m,n)]))*
       (-2*f2[ind(row+3,1,m,n)] + f1[ind(row+3,1,m,n)] + wave[ind(row+3,2,m,n)] -
       2*f2[ind(row+3,2,m,n)] + f1[ind(row+3,2,m,n)]) +
  (V[ind(row+3,1,m,n)]/(4*dx*dx))*
       (wave[ind(row+4,2,m,n)] - 2*wave[ind(row+3,2,m,n)] + wave[ind(row+2,2,m,n)] +
       f1[ind(row+4,1,m,n)] - 2*f1[ind(row+3,1,m,n)] + f1[ind(row+2,1,m,n)]));
	}
	
	//Applying Lower Right hand corner
	if(row==1 && col==1)
	{
		
		wave[ind(m-3,n-2,m,n)] = V[ind(m-3,n-2,m,n)]*dt*dx/(2*V[ind(m-3,n-2,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-4,n-2,m,n)]/dx + wave[ind(m-3,n-3,m,n)]/dx + 
       sqrt(2.)/(V[ind(m-3,n-2,m,n)]*dt)*f2[ind(m-3,n-2,m,n)]);

wave[ind(m-2,n-3,m,n)] = V[ind(m-2,n-3,m,n)]*dt*dx/(2*V[ind(m-2,n-3,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-3,n-3,m,n)]/dx + wave[ind(m-2,n-4,m,n)]/dx + 
       sqrt(2.)/(V[ind(m-2,n-3,m,n)]*dt)*f2[ind(m-2,n-3,m,n)]);

wave[ind(m-2,n-2,m,n)] = V[ind(m-2,n-2,m,n)]*dt*dx/(2*V[ind(m-2,n-2,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-3,n-2,m,n)]/dx + wave[ind(m-2,n-3,m,n)]/dx + 
       sqrt(2.)/(V[ind(m-2,n-2,m,n)]*dt)*f2[ind(m-2,n-2,m,n)]);
		
	}

	//Applying Lower Left hand corner
	if(row==5 && col==5)
	{
		wave[ind(m-3,1,m,n)] = V[ind(m-3,1,m,n)]*dt*dx/(2*V[ind(m-3,1,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-4,1,m,n)]/dx + wave[ind(m-3,2,m,n)]/dx +
       sqrt(2.)/(V[ind(m-3,1,m,n)]*dt)*f2[ind(m-3,1,m,n)]);

wave[ind(m-2,2,m,n)] = V[ind(m-2,2,m,n)]*dt*dx/(2*V[ind(m-2,2,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-3,2,m,n)]/dx + wave[ind(m-2,3,m,n)]/dx +
       sqrt(2.)/(V[ind(m-2,2,m,n)]*dt)*f2[ind(m-2,2,m,n)]);

wave[ind(m-2,1,m,n)] = V[ind(m-2,1,m,n)]*dt*dx/(2*V[ind(m-2,1,m,n)]*dt + sqrt(2.)*dx)*
       (wave[ind(m-3,1,m,n)]/dx + wave[ind(m-2,2,m,n)]/dx +
       sqrt(2.)/(V[ind(m-2,1,m,n)]*dt)*f2[ind(m-2,1,m,n)]);
	}
	

if(bc==1)
{

  // for upper right hand corner

  wave[ind(2,n-2,m,n)] = V[ind(2,n-2,m,n)]*dt*dx/(2*V[ind(2,n-2,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(3,n-2,m,n)]/dx + wave[ind(2,n-3,m,n)]/dx +
         sqrt(2.)/(V[ind(2,n-2,m,n)]*dt)*f2[ind(2,n-2,m,n)]); 

  wave[ind(1,n-3,m,n)] = V[ind(1,n-3,m,n)]*dt*dx/(2*V[ind(1,n-3,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(2,n-3,m,n)]/dx + wave[ind(1,n-4,m,n)]/dx +
         sqrt(2.)/(V[ind(1,n-3,m,n)]*dt)*f2[ind(1,n-3,m,n)]);

  wave[ind(1,n-2,m,n)] = V[ind(1,n-2,m,n)]*dt*dx/(2*V[ind(1,n-2,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(2,n-2,m,n)]/dx + wave[ind(1,n-3,m,n)]/dx +
         sqrt(2.)/(V[ind(1,n-2,m,n)]*dt)*f2[ind(1,n-2,m,n)]);

  // for upper left hand corner
  
  wave[ind(2,1,m,n)] = V[ind(2,1,m,n)]*dt*dx/(2*V[ind(2,1,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(3,1,m,n)]/dx + wave[ind(2,2,m,n)]/dx +
         sqrt(2.)/(V[ind(2,1,m,n)]*dt)*f2[ind(2,1,m,n)]);
  
  wave[ind(1,2,m,n)] = V[ind(1,2,m,n)]*dt*dx/(2*V[ind(1,2,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(2,2,m,n)]/dx + wave[ind(1,3,m,n)]/dx +
         sqrt(2.)/(V[ind(1,2,m,n)]*dt)*f2[ind(1,2,m,n)]); 

  wave[ind(1,1,m,n)] = V[ind(1,1,m,n)]*dt*dx/(2*V[ind(1,1,m,n)]*dt + sqrt(2.)*dx)*
         (wave[ind(2,1,m,n)]/dx + wave[ind(1,2,m,n)]/dx +
         sqrt(2.)/(V[ind(1,1,m,n)]*dt)*f2[ind(1,1,m,n)]);

}
	
}

__global__ void ABC_outer(float* d_velocity,float* d_field1, float* d_field2, float* d_wave_propagate_t, float dx, float dt, float bc,int m, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row >= m || col >= n)return;
	
	
	
}

__global__ void rickerWavelet(float* w, float f, float n, float dt)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int id=row*n+col;

	//float T=dt*(n-1);
	float t0=1/f;
	
	if(id>=n) return;

	float t=id*dt;
	float tau=t-t0;


	w[id]=(1-(tau*tau*f*f*PI*PI))*expf(-tau*tau*PI*f*PI*f);
	//if (row==1) printf("PI=%f\n",PI);

	

}

__global__ void extractCorrectRegion(float* A,float* B, int m, int n, int p)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= m || col >= n)return;

	A[ind(row,col,m,n)]=B[ind(row+p,col+p,m+2*p,n+2*p)];
}

__global__ void badBoundaryCondition(float* A, float* T, int m, int n, int p)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= m || col >= n)return;

	// First Zeroing matrix A
	T[ind(row+p,col+p,m+2*p,n+2*p)]=A[ind(row+p,col+p,m+2*p,n+2*p)];


}

__global__ void calculateCerjanCoeff(int p, float* G, int I)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int id=row*blockDim.y*gridDim.y+col;

	if(id>=p) return;

	float a=0.0049*(I-id);
	G[id]=expf(-a*a);

	// if(id<I-1) G[id]=1.0;
}

__global__ void cerjanMatrix(float* A, float* G, int m, int n, int p)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row >= m || col >= n)return;

	A[ind(row,col,m,n)]=1;

	if(row< p)	A[ind(row,col,m,n)]=A[ind(row,col,m,n)]*G[p-row-1];

	if(row>=m-p) A[ind(row,col,m,n)]=A[ind(row,col,m,n)]*G[row-(m-p)];

	if(col<p) A[ind(row,col,m,n)]=A[ind(row,col,m,n)]*G[p-col-1];

	if(col>=n-p) A[ind(row,col,m,n)]=A[ind(row,col,m,n)]*G[col-(n-p)];
	


}

__global__ void cerjanBoundaryCondition(float* A, float* B, float* C, float* CM, int m, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row >= m || col >= n)return;

	A[ind(row,col,m,n)]=A[ind(row,col,m,n)]*CM[ind(row,col,m,n)];
	B[ind(row,col,m,n)]=B[ind(row,col,m,n)]*CM[ind(row,col,m,n)];
	C[ind(row,col,m,n)]=C[ind(row,col,m,n)]*CM[ind(row,col,m,n)];


}